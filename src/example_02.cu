#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <cassert>
#include "example_02.h"

__global__ void kernel_vec_add(float *a, float *b, float *c, int n);

__host__ void vec_add(float *a, float *b, float *c, int n)
{
    float *dev_a, *dev_b, *dev_c;
    int size = sizeof(float) * n;

    hipMalloc(&dev_a, size);
    hipMalloc(&dev_b, size);
    hipMalloc(&dev_c, size);

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, size, hipMemcpyHostToDevice);
    
    kernel_vec_add<<<n / 32 + 1, 32>>>(dev_a, dev_b, dev_c, n);

    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}

__global__ void kernel_vec_add(float *a, float *b, float *c, int n)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n)
    {
        c[idx] = a[idx] + b[idx];
    }
}
