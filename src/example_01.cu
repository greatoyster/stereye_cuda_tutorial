#include <hip/hip_runtime.h>

#include <cstdio>

int main()
{
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n",
               static_cast<int>(error_id), hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    if (deviceCount == 0)
    {
        printf("There are no available device(s) that support CUDA\n");
    }
    else
    {
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    }

    for (int dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("Device %d: \"%s\"\n\n", dev, deviceProp.name);
        printf("Device warpSize: %d\n", deviceProp.warpSize);
        printf("Device maxGridSize: [%d, %d, %d]\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
        printf("Device maxThreadsDim: [%d, %d, %d]\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
        printf("Device maxThreadsPerBlock: %d\n", deviceProp.maxThreadsPerBlock);
        printf("Device multiProcessorCount: %d\n", deviceProp.multiProcessorCount);
        printf("Device maxBlocksPerMultiProcessor: %d\n", deviceProp.maxBlocksPerMultiProcessor);
    }
    return 0;
}